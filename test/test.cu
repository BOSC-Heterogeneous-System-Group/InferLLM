#include <hip/hip_runtime.h>

#include <time.h>
#include <algorithm>
#include <iostream>
#include <hip/hip_fp16.h>
using namespace std;

#define N  1024
// elementwise implementation copyed from https://github.com/Oneflow-Inc/oneflow/blob/master/oneflow/core/cuda/elementwise.cuh
constexpr int kBlockSize = 256;
constexpr int kNumWaves = 32;

inline hipError_t GetNumBlocks(int64_t n, int *num_blocks)
{
    int dev;
    {
        hipError_t err = hipGetDevice(&dev);
        if (err != hipSuccess)
        {
            return err;
        }
    }
    int sm_count;
    {
        hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
        if (err != hipSuccess)
        {
            return err;
        }
    }
    int tpm;
    {
        hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
        if (err != hipSuccess)
        {
            return err;
        }
    }
    *num_blocks = std::max<int>(1, std::min<int64_t>((n + kBlockSize - 1) / kBlockSize,
                                                     sm_count * tpm / kBlockSize * kNumWaves));
    return hipSuccess;
}

constexpr int kMaxPackBytes = 128 / 8;
constexpr int kMaxPackSize = 8;

template <typename Function, typename... Args>
__global__ void __launch_bounds__(kBlockSize)
    ApplyGeneric(Function functor, int64_t n, float *ret, Args... args)
{

    const int global_tid = blockIdx.x * kBlockSize + threadIdx.x;
    for (int64_t i = global_tid; i < n; i += blockDim.x * gridDim.x)
    {
        ret[i] = functor(i, args...);
    }
}

template <typename Function, typename... Args>
hipError_t LaunchKernel(Function fun, int64_t n, float *ret, Args... args)
{
    int num_blocks;
    {
        hipError_t err = GetNumBlocks(n,&num_blocks);
        if (err != hipSuccess)
        {
            return err;
        }
    }
    ApplyGeneric<<<num_blocks, kBlockSize>>>(fun, n, ret, args...);
    return hipPeekAtLastError();
}

struct MultiplyFunctor
{
    __device__ float operator()(uint32_t i, float *input1, float *input2) const
    {
        return input1[i] + input2[i];
    }
};

int main()
{
    float *x_host = (float *)malloc(N * sizeof(float));
    float *x_device;
    hipMalloc((void **)&x_device, N * sizeof(float));
    for (int i = 0; i < N; i++)
        x_host[i] = 2.0;
    hipMemcpy(x_device, x_host, N * sizeof(float), hipMemcpyHostToDevice);

    float *y_host = (float *)malloc(N * sizeof(float));
    float *y_device;
    hipMalloc((void **)&y_device, N * sizeof(float));
    for (int i = 0; i < N; i++)
        y_host[i] = 2.0;
    hipMemcpy(y_device, y_host, N * sizeof(float), hipMemcpyHostToDevice);

    float *output_host = (float *)malloc(N * sizeof(float));
    float *output_device;
    hipMalloc((void **)&output_device, N * sizeof(float));

    // naive elementwise
    int32_t block_num = (N + kBlockSize - 1) / kBlockSize;
    dim3 grid(block_num, 1);
    dim3 block(kBlockSize, 1);

    LaunchKernel(MultiplyFunctor(), N, output_device, x_device, y_device);
    hipMemcpy(output_host, output_device, N * sizeof(float), hipMemcpyDeviceToHost);

    // elementwise template

    for (int i = 0; i < N ; i++)
    {
        cout << output_host[i] << endl;
    }
    free(x_host);
    free(y_host);
    free(output_host);
    hipFree(x_device);
    hipFree(y_device);
    hipFree(output_device);
    return 0;
}
