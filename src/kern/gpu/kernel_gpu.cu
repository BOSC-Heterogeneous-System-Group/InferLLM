#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <vector>

#define ENABLE_GPU 1

#include "core/tensor.h"
#include "kern/kernel.h"
#include "kernel_gpu.h"
#include "math.h"
#include "string.h"
#include "utils.h"

namespace inferllm {
namespace gpu {

#define CUDA_KERNEL_LOOP(i, n)                                   \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
         i += blockDim.x * gridDim.x)

constexpr int kBlockSize = 256;
constexpr int kNumWaves = 32;
constexpr int DequantizedBlockSize = 256;
const int CUDA_NUM_THREADS = 512;

inline int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

__global__ void llm_elemwise_broadcast_dim0_src1_compute_float_add_gpu(
        const float* src0, const float* src1, float* dst, uint32_t len0,
        uint32_t len1) {
    int row = blockIdx.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < len1) {
        int index = row * len1 + col;
        dst[index] = src0[index] + src1[col];
    }
}

__global__ void llm_elemwise_broadcast_dim0_src1_compute_float_mul_gpu(
        const float* src0, const float* src1, float* dst, uint32_t rows,
        uint32_t ncols) {
    int row = blockIdx.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < ncols) {
        int index = row * ncols + col;
        dst[index] = src0[index] * src1[col];
    }
}

void llm_elemwise_broadcast_dim0_src1_compute_float(
        const float* src0, const float* src1, float* dst, uint32_t rows, uint32_t ncols,
        ElemMode mode, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(512, 1, 1);
    const dim3 block_nums((ncols + 511) / 512, rows, 1);
    switch (mode) {
        case ElemMode::Add: {
            llm_elemwise_broadcast_dim0_src1_compute_float_add_gpu<<<
                    block_nums, block_dims, 0, stream>>>(src0, src1, dst, rows, ncols);
            break;
        }
        case ElemMode::Mul: {
            llm_elemwise_broadcast_dim0_src1_compute_float_mul_gpu<<<
                    block_nums, block_dims, 0, stream>>>(src0, src1, dst, rows, ncols);
            break;
        }
        default:
            INFER_ASSERT(0, "Not supported.");
    }
}


__global__ void softmax_f32_cuda(const float* x, float* dst, const int cols) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const float* src = x + row * cols;
    dst = dst + row * cols;

    float max = -INFINITY;
    for (int col = tid; col < cols; col += block_size) {
        const float val = src[col];
        max = val > max ? val : max;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        float temp = __shfl_xor_sync(0xffffffff, max, mask);
        max = max > temp ? max : temp;
    }

    float sum = 0.0;
    for (int col = tid; col < cols; col += block_size) {
        const float val = expf(src[col] - max);
        sum += val;
        dst[col] = val;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        sum += __shfl_xor_sync(0xffffffff, sum, mask, 32);
    }

    for (int col = tid; col < cols; col += block_size) {
        dst[col] /= sum;
    }
}

void llm_softmax_compute_float(
        const float* src, float* dst, uint32_t len_row, uint32_t col,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(kNumWaves, 1, 1);
    const dim3 block_nums(1, len_row, 1);
    softmax_f32_cuda<<<block_nums, block_dims, 0, stream>>>(src, dst, col);
}

__global__ void embeding_float_cuda(
        const float* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd) {
    int seq_id = blockIdx.y;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < embd) {
        uint32_t row = index[seq_id];
        dst = dst + seq_id * embd;
        weights = weights + row * embd;
        dst[thread_id] = weights[thread_id];
    }
}

void llm_embedding_get_float_float(
        const float* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(512, 1, 1);
    const dim3 block_nums((embd + 512) / 512, len_seq, 1);
    embeding_float_cuda<<<block_nums, block_dims, 0, stream>>>(
            weights, index, dst, len_seq, embd);
}

__global__ void llm_embedding_get_int4_float_gpu(
        const void* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, const int weight_stride) {
    int seq_id = blockIdx.y;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < embd / 2) {
        uint32_t row = index[seq_id];
        dst = dst + seq_id * embd;
        const void* src = (static_cast<const char*>(weights) + row * weight_stride);
        int q40_block_id = thread_id * 2 / QK40;
        int block_offset = thread_id % (QK40 / 2);
        BlockQ40* q40_block = (BlockQ40*)src + q40_block_id;
        float scale = q40_block->d;
        uint8_t value = q40_block->qs[block_offset];
        const int8_t v1 = value & 0xf;
        const int8_t v2 = value >> 4;
        dst[thread_id * 2] = (v1 - 8) * scale;
        dst[thread_id * 2 + 1] = (v2 - 8) * scale;
    }
}

void llm_embedding_get_int4_float(
        const void* weights, const uint32_t* index, float* dst, uint32_t len_seq,
        uint32_t embd, cudaHandle* handle) {
    const int weight_stride = embd * sizeof(BlockQ40) / QK40;
    // one thread compute two data
    int grid_1 = (embd / 2 + DequantizedBlockSize - 1) / DequantizedBlockSize;
    dim3 grid(grid_1, len_seq);
    hipStream_t stream = handle->stream;
    llm_embedding_get_int4_float_gpu<<<grid, DequantizedBlockSize, 0, stream>>>(
            weights, index, dst, len_seq, embd, weight_stride);
}

struct SiluFunctor {
    __device__ float operator()(uint32_t i, const float* input) const {
        float src = input[i];
        return src / (1.0 + exp(-src));
    }
};

struct GeluFunctor {
    __device__ float operator()(uint32_t i, const float* input) const {
        float src = input[i];
        return 0.5 * src * (1 + tanh(sqrt(2.0 / PI) * (src + PGELU * src * src * src)));
    }
};

struct AddFunctor {
    __device__ float operator()(
            uint32_t i, const float* input1, const float* input2) const {
        return input1[i] + input2[i];
    }
};

struct MulFunctor {
    __device__ float operator()(
            uint32_t i, const float* input1, const float* input2) const {
        return input1[i] * input2[i];
    }
};

template <typename Function, typename... Args>
__global__ void ApplyFunction(Function functor, int64_t n, float* ret, Args... args) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        ret[tid] = functor(tid, args...);
    }
}

template <typename Function, typename... Args>
hipError_t LaunchKernel(
        Function fun, hipStream_t stream, int64_t n, float* ret, Args... args) {
    int num_blocks = (n + kBlockSize - 1) / kBlockSize;
    ApplyFunction<Function, Args...>
            <<<num_blocks, kBlockSize, 0, stream>>>(fun, n, ret, args...);
    return hipPeekAtLastError();
}

void llm_elemwise_compute_float(
        InData<float> srcs, float* dst, size_t len, ElemMode mode, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    switch (mode) {
        case ElemMode::Add: {
            const float* src0 = srcs[0];
            const float* src1 = srcs[1];
            LaunchKernel(AddFunctor(), stream, len, dst, src0, src1);
            break;
        }
        case ElemMode::Mul: {
            const float* src0 = srcs[0];
            const float* src1 = srcs[1];

            LaunchKernel(MulFunctor(), stream, len, dst, src0, src1);
            break;
        }
        case ElemMode::Silu: {
            const float* src0 = srcs[0];
            LaunchKernel(SiluFunctor(), stream, len, dst, src0);
            break;
        }
        case ElemMode::Gelu: {
            const float* src0 = srcs[0];
            LaunchKernel(GeluFunctor(), stream, len, dst, src0);
            break;
        }
        default:
            INFER_ASSERT(0, "Not supported.");
    }
}

__global__ void rms_norm_f32(const float* x, float* dst, const int ncols, float eps) {
    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    const int WARP_SIZE = blockDim.x;

    float tmp = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    const float mean = tmp / ncols;
    const float scale = 1.0f / sqrtf(mean + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row * ncols + col] = scale * x[row * ncols + col];
    }
}

void llm_rms_norm_compute_float(
        const float* src, float* dst, uint32_t seq_len, uint32_t embd, float eps,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    rms_norm_f32<<<seq_len, kNumWaves, 0, stream>>>(
            src, dst,  embd, eps);
}

__global__ void norm_f32(const float* x, float* dst, const int ncols, float eps) {
    const int row = blockIdx.x;
    const int tid = threadIdx.x;
    const int WARP_SIZE = blockDim.x;

    float mean = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col];
        mean += xi;
    }
    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        mean += __shfl_xor_sync(0xffffffff, mean, mask, 32);
    }
    mean = mean / ncols;

    float sum = 0.0f;  // partial sum for thread in warp
    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row * ncols + col] - mean;
        sum += xi * xi;
        dst[row * ncols + col] = xi;
    }
    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        sum += __shfl_xor_sync(0xffffffff, sum, mask, 32);
    }
    const float scale = 1.0f / sqrtf(sum / ncols + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row * ncols + col] = scale * x[row * ncols + col];
    }
}

void llm_norm_compute_float(
        const float* src, float* dst, uint32_t seq_len, uint32_t embd, float eps,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    norm_f32<<<seq_len, kNumWaves, 0, stream>>>(src, dst, embd, eps);
}

template <bool halfmode>
__global__ void rope_compute_float(
        float* dst, const float* src, float theta_scale, uint32_t position_offset,
        uint32_t n_rot, uint32_t seqlen, uint32_t n_head, uint32_t head_embd) {
    const int rot = threadIdx.x;
    const int head = blockIdx.x;
    const int seq = blockIdx.y;

    if (rot >= n_rot / 2 || head >= n_head || seq >= seqlen) {
        return;
    }

    const float theta = (position_offset + seq) * powf(theta_scale, rot);
    const float sin_theta = sinf(theta);
    const float cos_theta = cosf(theta);

    const int offset = seq * n_head * head_embd + head * head_embd;
    if (halfmode) {
        const int half_embd = head_embd / 2;
        const float x0 = src[offset + rot];
        const float x1 = src[offset + rot + half_embd];
        dst[offset + rot] = x0 * cos_theta - x1 * sin_theta;
        dst[offset + rot + half_embd] = x0 * sin_theta + x1 * cos_theta;
    } else {
        const float x0 = src[offset + 2 * rot];
        const float x1 = src[offset + 2 * rot + 1];
        dst[offset + 2 * rot] = x0 * cos_theta - x1 * sin_theta;
        dst[offset + 2 * rot + 1] = x0 * sin_theta + x1 * cos_theta;
    }
}

void llm_rope_compute_float(
        float* dst, const float* src, uint32_t n_past, uint32_t n_rot, RotMode m,
        uint32_t seqlen, uint32_t head, uint32_t head_embd, cudaHandle* handle) {
    hipStream_t stream = handle->stream;

    const float theta_scale = powf(10000.0, -2.0f / n_rot);
    const float position_offset = n_past;

    INFER_ASSERT(n_rot <= 2048, "n_rot is two large.");
    INFER_ASSERT(n_rot % 2 == 0, "n_rot must be even.");
    const dim3 block_dims(n_rot / 2, 1, 1);
    const dim3 block_nums(head, seqlen, 1);
    //! offset to nr_past
    if (m == RotMode::Mode1) {
        src = src + n_past * head_embd * head;
        dst = dst + n_past * head_embd * head;
    }
    if (m == RotMode::ModelRotHalf) {
        rope_compute_float<true><<<block_nums, block_dims, 0, stream>>>(
                dst, src, theta_scale, position_offset, n_rot, seqlen, head, head_embd);
    } else {
        rope_compute_float<false><<<block_nums, block_dims, 0, stream>>>(
                dst, src, theta_scale, position_offset, n_rot, seqlen, head, head_embd);
    }
}

__global__ void llm_elemwise_compute_float_scale_gpu(
        float* src, float* dst, size_t len, float scale) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        dst[index] = src[index] * scale;
    }
}

void llm_elemwise_compute_float_scale(
        float* src, float* dst, size_t len, float scale, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    const dim3 block_dims(CUDA_NUM_THREADS, 1, 1);
    const dim3 block_nums((len + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS, 1, 1);
    llm_elemwise_compute_float_scale_gpu<<<block_nums, block_dims, 0, stream>>>(
            src, dst, len, scale);
}

void llm_matmul_compute_float_float(
        float* dst, const float* src0, const float* bias, const float* src1, uint32_t M,
        uint32_t N, uint32_t K, void* workspace, uint32_t size, cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemm(
            cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K,
            &alpha, src0, K, src1, K, &beta, dst, N));
    if (bias != nullptr) {
        llm_elemwise_broadcast_dim0_src1_compute_float(
                dst, bias, dst, M, N, ElemMode::Add, handle);
    }
}

__global__ void dequantize_mul_mat_vec(
        const void* dx, const float* y, const float* bias, float* dst, const int M,
        const int N, const int K) {
        const int m_id = blockIdx.y;
    const int n_id = blockIdx.x * blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    if (m_id >= M || n_id >= N) {
        return;
    }

    const int iter_stride = 2 * 32;
    const int vals_per_iter =
            iter_stride / 32;  // num quantized vals per thread and i iter

    // partial sum for each thread
    float tmp = 0.0f;
    const float* srcy = y + m_id * K;
    dst = dst + m_id * N;
    float bias_val = bias ? bias[n_id] : 0.0f;

    for (int i = 0; i < K; i += iter_stride) {
        const int col = i + vals_per_iter * tid;
        const int ib = (n_id * K + col) / QK40;  // x block index
        const int iqs = (col % QK40) / 2;        // x quant index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights
            // per data val
            float2 v;
            const BlockQ40* x = (const BlockQ40*)dx + ib;
            const float d = x->d;
            const int vui = x->qs[iqs];

            v.x = vui & 0xF;
            v.y = vui >> 4;

            v.x = (v.x - 8.0f) * d;
            v.y = (v.y - 8.0f) * d;

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of
            // y_offset = qk/2
            tmp += v.x * srcy[col];
            tmp += v.y * srcy[col + 1];
        }
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[n_id] = tmp + bias_val;
    }
}

void llm_matmul_compute_int4_float(
        float* dst, const void* src0, const float* bias, const float* src1, uint32_t M,
        uint32_t N, uint32_t K, void* workspace, uint32_t size, cudaHandle* handle) {
    INFER_ASSERT(K % QK40 == 0, "embd is not the time of QK40.");
    hipStream_t stream = handle->stream;
    const dim3 block_nums(N + 15 / 16, M, 1);
    const dim3 block_dims(32, 16, 1);
    dequantize_mul_mat_vec<<<block_nums, block_dims, 0, stream>>>(
            src0, src1, bias, dst, M, N, K);
}

__global__ void llm_scale_diag_mask_inf_float_gpu(
        const float* src, float* dst, const int past, const int len, const int head_dim,
        float scale) {
    const int head = blockIdx.z;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= past + len || row >= len || head >= head_dim)
        return;

    const int row_stride = len + past;
    const int head_stride = len * (len + past);

    src = src + head * head_stride + row * row_stride;
    dst = dst + head * head_stride + row * row_stride;

    dst[col] = (col > past + row) ? -INFINITY : src[col] * scale;
}

void llm_scale_diag_mask_inf_float(
        float* dst, const float* src, float scale, uint32_t past, uint32_t seqlen,
        uint32_t head, cudaHandle* handle) {
    hipStream_t stream = handle->stream;

    constexpr int kBlockSize = 32;
    const int block_y = (seqlen + kBlockSize - 1) / kBlockSize;
    const int block_x = (past + seqlen + kBlockSize - 1) / kBlockSize;
    const dim3 block_dims(kBlockSize, kBlockSize, 1);
    const dim3 block_nums(block_x, block_y, head);

    llm_scale_diag_mask_inf_float_gpu<<<block_nums, block_dims, 0, stream>>>(
            src, dst, past, seqlen, head, scale);
}

__global__ void diag_mask_inf_f32(
        const float* src, float* dst, const int past, const int len,
        const int head_dim) {
    const int head = blockIdx.z;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= len || row >= len || head >= head_dim)
        return;

    const int row_stride = len + past;
    const int head_stride = len * (len + past);

    src = src + head * head_stride + row * row_stride + past;
    dst = dst + head * head_stride + row * row_stride + past;
    dst[col] = (col > row) ? -INFINITY : src[col];
}

void llm_diag_mask_inf_float(
        float* dst, const float* src, uint32_t n_past, uint32_t N, uint32_t head,
        cudaHandle* handle) {
    hipStream_t stream = handle->stream;
    constexpr int kBlockSize = 32;
    const int block_n = (N + kBlockSize - 1) / kBlockSize;
    const dim3 block_dims(kBlockSize, kBlockSize, 1);
    const dim3 block_nums(block_n, block_n, head);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(src, dst, n_past, N, head);
}

void llm_permute_compute_float(
        float* dst, const float* src0, uint32_t dim0, uint32_t dim1, uint32_t dim2,
        std::vector<uint32_t> param, cudaHandle* handle) {
    return;
}
/**
 * dst :head *seqlen *(seql)
 */

void llm_matmul_compute_with_head_stride_float(
        float* dst, const float* srck, const float* srcq, uint32_t seqlen,
        uint32_t embd, uint32_t head, uint32_t nr_past, cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = seqlen;
    uint32_t N = seqlen + nr_past;
    uint32_t K = head_embd;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;
    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemmStridedBatched(
            cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, srck, embd,
            head_embd, srcq, embd, head_embd, &beta, dst, N, M * N, head));
}

void llm_head_batched_matmul_compute_float(
        float* dst, const float* v, const float* qk, uint32_t seqlen, uint32_t embd,
        uint32_t head, uint32_t nr_past, cudaHandle* handle) {
    uint32_t head_embd = embd / head;
    uint32_t M = head_embd;
    uint32_t K = seqlen + nr_past;
    uint32_t N = seqlen;
    hipStream_t stream = handle->stream;
    hipblasHandle_t cublas_handle = handle->cublas_handle;
    float alpha = 1.f;
    float beta = 0.f;

    CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));
    CUBLAS_CHECK(hipblasSgemmStridedBatched(
            cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, v, embd,
            head_embd, qk, K, K * N, &beta, dst, embd, head_embd, head));
}

void llm_glm_gmask_inf_float(
        float* dst, uint32_t n_past, uint32_t seqlen, uint32_t head,
        cudaHandle* handle) {
    //! set every head the last number of data to -inf of every row expect
    //! the
    //! last row
    // const int nc = n_past + seqlen;
    // auto task = [=](const TaskId& id) {
    //     for (int k = id.start; k < id.end; k++) {
    //         for (int j = 0; j < seqlen - 1; j++) {
    //             dst[k * nc * seqlen + j * nc + nc - 1] = -INFINITY;
    //         }
    //     }
    // };
}
void llm_glm_rope_compute_float(
        float* dst, const float* src0, uint32_t n_past, uint32_t gmask_positon,
        uint32_t seqlen, uint32_t head, uint32_t embd, cudaHandle* handle) {
    // bool prefill = false;
    // if (n_past == 0) {
    //     prefill = true;
    // }
    // int quart_embd = embd / 4;
    // int half_embd = embd / 2;
    // auto task = [=](const TaskId& id) {
    //     for (int h = id.start; h < id.end; h++) {
    //         for (int seq = 0; seq < seqlen; seq++) {
    //             int position_id = std::min(seq + n_past, gmask_positon);
    //             int block_position_id =
    //                     std::max((int)(n_past + seq) - (int)gmask_positon, 0);
    //             for (int p = 0; p < quart_embd; p++) {
    //                 const double theta = pow(10000.0, ((double)-2 * p) /
    //                 (half_embd)); const double cos_theta = cos(position_id * theta);
    //                 const double sin_theta = sin(position_id * theta);

    //                 const double cos_theta_b = cos(block_position_id * theta);
    //                 const double sin_theta_b = sin(block_position_id * theta);

    //                 //! first half
    //                 {
    //                     const float* const src =
    //                             src0 + seq * head * embd + h * embd + p;
    //                     float* dst_data = dst + seq * head * embd + h * embd + p;
    //                     double x0 = src[0];
    //                     double x32 = src[quart_embd];
    //                     dst_data[0] = x0 * cos_theta - x32 * sin_theta;
    //                     dst_data[quart_embd] = x32 * cos_theta + x0 * sin_theta;
    //                 }
    //                 //! second half
    //                 {
    //                     const float* const src =
    //                             src0 + seq * head * embd + h * embd + half_embd + p;
    //                     float* dst_data =
    //                             dst + seq * head * embd + h * embd + half_embd + p;
    //                     double x0 = src[0];
    //                     double x32 = src[quart_embd];
    //                     dst_data[0] = x0 * cos_theta_b - x32 * sin_theta_b;
    //                     dst_data[quart_embd] = x32 * cos_theta_b + x0 * sin_theta_b;
    //                 }
    //             }
    //         }
    //     }
    // };
}

void llm_matmul_compute_with_head_strideq_broadcastk_float(
        float* dst, const float* srck, const float* srcq, uint32_t seqlen,
        uint32_t embd, uint32_t head, uint32_t query_group_num, uint32_t nr_past,
        cudaHandle* handle) {}

void llm_head_batched_matmul_broadcastv_float(
        float* dst, const float* v, const float* qk, uint32_t seqlen, uint32_t embd,
        uint32_t head, uint32_t query_group_num, uint32_t nr_past, cudaHandle* handle) {
}

size_t llm_matmul_get_workspace_float(uint32_t M, uint32_t N, uint32_t K) {
    return 0;
}

size_t llm_matmul_get_workspace_float_float(uint32_t M, uint32_t N, uint32_t K) {
    return 0;
}
}  // namespace gpu
}  // namespace inferllm